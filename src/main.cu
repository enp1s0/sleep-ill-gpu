#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <unistd.h>

int main(int argc, char** argv) {
	if (argc < 4) {
		std::printf("Usage : %s [sleep duration (s)] [valid min clock (MHz)] [valid max clock (MHz)]\n", argv[0]);
		return 1;
	}

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	const auto sleep_duration = std::stoi(argv[1]);
	const auto min_clock = std::stoi(argv[2]);
	const auto max_clock = std::stoi(argv[3]);

	const auto freq_in_MHz = prop.clockRate / 1000;
	const auto ill_state = (freq_in_MHz < min_clock) || (freq_in_MHz > max_clock);


	std::printf("[%s] SM Frequency : %d MHz (state=%s)\n",
			prop.name,
			freq_in_MHz,
			ill_state ? "ill" : "good"
			);
	std::fflush(stdout);

	if (ill_state) {
		std::printf("Sleep %d sec\n", sleep_duration);
		std::fflush(stdout);
		sleep(sleep_duration);
	}
}
