#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <nvml.h>
#include <unistd.h>

int main(int argc, char** argv) {
	if (argc < 4) {
		std::printf("Usage : %s [sleep duration (s)] [valid min clock (MHz)] [valid max clock (MHz)]\n", argv[0]);
		return 1;
	}

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	const unsigned sleep_duration = std::stoi(argv[1]);
	const unsigned min_clock = std::stoi(argv[2]);
	const unsigned max_clock = std::stoi(argv[3]);

	nvmlDevice_t device;

	nvmlInit();
	nvmlDeviceGetHandleByIndex(0, &device);

	unsigned freq_in_MHz;
	const auto res = nvmlDeviceGetClock(device, NVML_CLOCK_SM, NVML_CLOCK_ID_CURRENT, &freq_in_MHz);
	if (res != NVML_SUCCESS) {
		std::printf("Error at nvmlDeviceGetClock (error code = %u)\n",
				static_cast<unsigned>(res));
		return 1;
	}
	const auto ill_state = (freq_in_MHz < min_clock) || (freq_in_MHz > max_clock);


	std::printf("[%s] SM Frequency : %u MHz (state=%s)\n",
			prop.name,
			freq_in_MHz,
			ill_state ? "ill" : "good"
			);
	std::fflush(stdout);

	if (ill_state) {
		std::printf("Sleep %u sec\n", sleep_duration);
		std::fflush(stdout);
		sleep(sleep_duration);
	}
}
